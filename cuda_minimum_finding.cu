
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 8000000

__device__ int findMinimum(int a[], int low, int high)
{
    int min = a[low];
    for (int i = low; i < high; ++i)
    {
        if (min > a[i])
        {
            min = a[i];
        }
    }
    return min;
}

__global__ void min(int *a, int *c)
{
    int rank = threadIdx.x;
    int p = sizeof(c)/sizeof(int);
    int numToSort = (8 * 1000000) / p;
    int low = rank * numToSort;
    int high = low + numToSort - 1;
    c[rank] = findMinimum(a, low, high);
    printf("crash here 2.5 \n");
}

int main()
{
    int num_threads = 8;
    int a[N];
    int *dev_a;
    int c[num_threads];
    int *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, num_threads * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        a[i] = rand() % 1000000000;
    }
    printf("crash here 1 \n");
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, num_threads * sizeof(int), hipMemcpyHostToDevice);
    printf("crash here 2 \n");
    dim3 grid(1);
    dim3 threads(num_threads);
    min <<<grid, threads >>> (dev_a, dev_c);
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("crash here 3 \n");
    hipDeviceSynchronize();  // Waits for threads to finish
    int min = c[0];
    for(int i = 0; i < num_threads; ++i){
        if(min > c[i]){
            min = c[i];
            printf("Minimal value parallel with cuda is: %d\n", min);
        }
    }
    return 0;
}
