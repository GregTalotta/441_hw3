#include "hip/hip_runtime.h"
#include "stdio.h"

#define N 8 * 1000000

__global__ void hello()
{
    printf("hello, world from the GPU\n");
}

int main()
{
    int a[N];
    int *dev_a;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    for (i = 0; i < N; i++)
    {
        a[i] = rand() % 1000000000;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    
    dim3 grid(1);
    dim3 threads(8);
    hello <<<grid, threads >>> (dev_a, );
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();  // Waits for threads to finish
    return 0;
}
