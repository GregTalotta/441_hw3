#include "hip/hip_runtime.h"
#include "stdio.h"

#define N 8 * 1000000

_device_ int findMinimum(int a[], int low, int high)
{
    int min = a[low];
    for (int i = low; i < high; ++i)
    {
        if (min > a[i])
        {
            min = a[i];
        }
    }
    return min;
}

__global__ void min(int a[], int c[])
{
    int rank = threadIdx.x;
    int p = sizeof(dev_c)/sizeof(int);
    int numToSort = (8 * 1000000) / p;
    int low = rank * numToSort;
    int high = low + numToSort - 1;
    int c[rank] = findMinimum(dev_a, low, high);
}

int main()
{
    int num_threads = 8;
    int a[N];
    int *dev_a;
    int c[num_threads];
    int *dev_c;
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, num_threads * sizeof(int));
    for (i = 0; i < N; i++)
    {
        a[i] = rand() % 1000000000;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, num_threads * sizeof(int), hipMemcpyHostToDevice);
    dim3 grid(1);
    dim3 threads(num_threads);
    min <<<grid, threads >>> (dev_a, dev_c);
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();  // Waits for threads to finish
    int min = c[0];
    for(int i = 0; i < num_threads; ++i){
        if(min > c[i]){
            min = c[i];
            printf("Minimal value parallel with cuda is: %d\n", min);
        }
    }
    return 0;
}
