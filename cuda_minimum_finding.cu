
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

#define N 8000000
#define T 8

__global__ void find_min(int *a, int *c)
{
    int rank = threadIdx.x;
    int numToSort = (8000000) / T;
    int low = rank * numToSort;
    int high = low + numToSort - 1;
    int min = a[low];
    for (int i = low; i < high; ++i)
    {
        if (min > a[i])
        {
            min = a[i];
        }
    }
    c[rank] = min;
    printf("crash here 2.5 \n");
}

int main()
{
    printf("start\n");
    int *a;
    a = (int *)malloc(sizeof(int) * N);
    int *dev_a;
    int c[T];
    int *dev_c;
    printf("crash here 0 \n");
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, T * sizeof(int));
    for (int i = 0; i < N; ++i)
    {
        a[i] = rand() % 1000000000;
    }
    for(int i = 0; i < T; ++i){
        c[i] = 1000000001;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, T * sizeof(int), hipMemcpyHostToDevice);
    dim3 grid(1);
    find_min <<<grid, T >>> (dev_a, dev_c);
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();  // Waits for threads to finish
    int min = c[0];
    for(int i = 0; i < T; ++i){
        if(min > c[i]){
            min = c[i];
        }
    }
    printf("Minimal value parallel with cuda is: %d\n", min);
    min = a[0];
        for(int i =0; i < N; ++i){
            if(min > a[i]){
                min = a[i];
            }
        }
        printf("Minimal value sequential: %d\n", min);
    return 0;
}


