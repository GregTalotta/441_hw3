
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 8000000

__global__ void find_min(int *a, int *c)
{
    int rank = threadIdx.x;
    int p = sizeof(c)/sizeof(int);
    int numToSort = (8 * 1000000) / p;
    int low = rank * numToSort;
    int high = low + numToSort - 1;
    int min = a[low];
    for (int i = low; i < high; ++i)
    {
        if (min > a[i])
        {
            min = a[i];
        }
    }
    c[rank] = min;
    printf("crash here 2.5 \n");
}

int main()
{
    printf("start\n");
    int num_threads = 8;
    int a[N];
    int *dev_a;
    int c[num_threads];
    int *dev_c;

    printf("crash here 0 \n");
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_c, num_threads * sizeof(int));
    //fill the array
    for (int i = 0; i < N; i++)
    {
        a[i] = rand() % 1000000000;
    }

    printf("crash here 1 \n");
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    printf("crash here 2 \n");
    dim3 grid(1);
    dim3 threads(num_threads);
    find_min <<<grid, threads >>> (dev_a, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("crash here 3 \n");
    hipDeviceSynchronize();  // Waits for threads to finish
    int min = c[0];
    for(int i = 0; i < num_threads; ++i){
        if(min > c[i]){
            min = c[i];
            printf("Minimal value parallel with cuda is: %d\n", min);
        }
    }
    return 0;
}


