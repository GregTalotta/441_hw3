
#include <hip/hip_runtime.h>
#include "stdio.h"
#define DIM 8

__global__ void add(int* a, int* c)
{
    int row = blockIdx.x;
    int column = threadIdx.x;
    c[row] += a[(DIM*row)+column];
}

int main()
{
    int a[DIM][DIM], c[DIM];
    int* dev_a, * dev_c;

    hipMalloc((void**)&dev_a, DIM * DIM * sizeof(int));
    hipMalloc((void**)&dev_c, DIM * sizeof(int));

    for (int y = 0; y < DIM; y++)              // Fill Arrays
        for (int x = 0; x < DIM; x++)
            a[y][x] = 7;

    for(int i = 0; i < DIM; ++i){
        c[i]=0;
    }
    hipMemcpy(dev_a, a, DIM * DIM * sizeof(int), hipMemcpyHostToDevice);

    add <<<DIM, DIM >>> (dev_a, dev_c);

    hipMemcpy(c, dev_c, DIM * sizeof(int), hipMemcpyDeviceToHost);
    int total = 0;
    for(int i = 0; i < DIM; ++i){
        total += c[i];
        printf("c is: %d\n", c[i]);
    }
    printf("Total sum of all elements is: %d\n", total);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}
                 