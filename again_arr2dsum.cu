#include "hip/hip_runtime.h"
#include "stdio.h"
#define DIM 8
const int THREADS_PER_BLOCK = 8;
const int NUM_BLOCKS = 8; 

__global__ void add(int* a, int* c)
{
    __shared__ int chache[THREADS_PER_BLOCK];
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);
    int cacheIndex = threadIdx.x;
    int temp = 0;
    temp = a[tid];
    cache[cacheIndex] = temp;

    int i = blockDim.x / 2;
    while (i > 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (threadIdx.x == 0)         // if at thread 0 in this block
        c[blockIdx.x] = cache[0]; // save the sum in global memory

}

int main()
{
    int a[DIM][DIM], c[DIM];
    int* dev_a, * dev_c;

    hipMalloc((void**)&dev_a, DIM * DIM * sizeof(int));
    hipMalloc((void**)&dev_c, DIM * sizeof(int));

    for (int y = 0; y < DIM; y++)              // Fill Arrays
        for (int x = 0; x < DIM; x++)
            a[y][x] = 7;

    for(int i = 0; i < DIM; ++i){
        c[i]=0;
    }
    hipMemcpy(dev_a, a, DIM * DIM * sizeof(int), hipMemcpyHostToDevice);

    
    add <<<NUM_BLOCKS, THREADS_PER_BLOCK >>> (dev_a, dev_c);

    hipMemcpy(c, dev_c, DIM * sizeof(int), hipMemcpyDeviceToHost);
    int total = 0;
    for(int i = 0; i < DIM; ++i){
        total += c[i];
        printf("c is: %d\n", c[i]);
    }
    printf("Total sum of all elements is: %d\n", total);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}
