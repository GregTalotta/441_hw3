
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 4
#define ROWS 3

__global__ void add(int* a, int* c)
{
    int column = blockIdx.x;
    int total = 0;
    for(int i = 0; i < ROWS; ++i){
        total += a[COLUMNS*i + column];
    }
    c[column]=total;
}

int main()
{
    int a[ROWS][COLUMNS], c[COLUMNS];
    int* dev_a, * dev_c;

    hipMalloc((void**)&dev_a, ROWS * COLUMNS * sizeof(int));
    hipMalloc((void**)&dev_c, COLUMNS * sizeof(int));

    for (int y = 0; y < ROWS; y++)              // Fill Arrays
        for (int x = 0; x < COLUMNS; x++)
            a[y][x] = 7;

    hipMemcpy(dev_a, a, ROWS * COLUMNS * sizeof(int), hipMemcpyHostToDevice);

    add <<<1, COLUMNS >>> (dev_a, dev_c);

    hipMemcpy(c, dev_c, COLUMNS * sizeof(int), hipMemcpyDeviceToHost);
    int total = 0;
    for(int i = 0; i < COLUMNS; ++i){
        total += c[0];
    }
    printf("Total sum of all elements is: %d\n", total);
    hipFree(dev_a);
    hipFree(dev_c);
    return 0;
}
                 